#include "hip/hip_runtime.h"
#include "include/bce_cost.hh"
#include <cmath>
#include <cassert>
#include "include/nn_exception.hh"

__global__ void binaryCrossEntropyCost(float* predictions, float* target, int size, float* cost) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        float pred = predictions[index];
        pred = fmaxf(fminf(pred, 1.0f - 1e-7), 1e-7);

        float partial_cost = target[index] * logf(pred)
                + (1.0f - target[index]) * logf(1.0f - pred);
        atomicAdd(cost, - partial_cost / size);
    }
}

__global__ void dBinaryCrossEntropyCost(float* predictions, float* target, float* dY,
                                        int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        // Clamp predictions to avoid division by zero
        float pred = predictions[index];
        pred = fmaxf(fminf(pred, 1.0f - 1e-7), 1e-7);

        dY[index] = -1.0 * (target[index] / pred - (1 - target[index]) / (1 - pred));
    }
}

//cost (or loss)
float BCECost::cost(Matrix predictions, Matrix target) {
 assert(predictions.shape.x == target.shape.x);

 float* cost;
 hipMallocManaged(&cost, sizeof(float));
 *cost = 0.0f;

 dim3 block_size(256);
 dim3 num_of_blocks((predictions.shape.x + block_size.x - 1) / block_size.x);
 binaryCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device.get(),
                target.data_device.get(),
                predictions.shape.x, cost);
 hipDeviceSynchronize();
 NNException::throwIfDeviceErrorsOccurred("Cannot compute binary cross entropy cost.");

 float cost_value = *cost;
 hipFree(cost);

 return cost_value;
}

//derivative of cost (aka derivative of loss)
Matrix BCECost::dCost(Matrix predictions, Matrix target, Matrix dY) {
 assert(predictions.shape.x == target.shape.x);

 dim3 block_size(256);
 dim3 num_of_blocks((predictions.shape.x + block_size.x - 1) / block_size.x);
 dBinaryCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device.get(),
                 target.data_device.get(),
                 dY.data_device.get(),
                 predictions.shape.x);
 NNException::throwIfDeviceErrorsOccurred("Cannot compute derivative for binary cross entropy.");

 return dY;
}