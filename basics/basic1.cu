
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){
    printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

__host__ int main(){
    hello<<<3, 3>>>();
    hipDeviceSynchronize();
}